#include "hip/hip_runtime.h"
#include <iostream>
#include "Tuple.h"
#include "Canvas.h"
#include "Matrix.h"
#include "Transformation.h"
#include "Sphere.h"
#include "Intersection.h"
#include <filesystem>
#include <cmath>

#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>

struct Projectile
{
    Tuple position;
    Tuple velocity;
    Projectile(const Tuple &pos, const Tuple &vel) : position(pos), velocity(vel) {}
};

struct Environment
{
    Tuple gravity;
    Tuple wind;
    Environment(const Tuple &gravity, const Tuple &wind) : gravity(gravity), wind(wind) {}
};

void tick(Projectile* proj, Environment* env) {
    proj->position += proj->velocity;
    proj->velocity += env->gravity + env->wind;
}

void challenge_projectile(){
    Canvas canvas(900, 550);

    Tuple initialPosition = Tuple::vector(0, 1, 0);
    Tuple initialVelocity = Tuple::normalize(Tuple::vector(1, 1.8, 0)) * 11.25;
    Projectile proj(initialPosition, initialVelocity);

    Tuple gravity = Tuple::vector(0, -0.1, 0);
    Tuple wind = Tuple::vector(-0.01, 0, 0);
    Environment env(gravity, wind );


    while (proj.position.y >= 0){
        tick(&proj, &env);
        std::cout << "Projectile Pos: " << proj.position << " Projectile Vel: " << proj.velocity << std::endl;
        try{
            canvas.WritePixel(proj.position.x, canvas.height - proj.position.y, Tuple::color(1, 1, 1, 1));
            canvas.WritePixel(proj.position.x+1, canvas.height - proj.position.y, Tuple::color(1, 1, 1, 1));
            canvas.WritePixel(proj.position.x-1, canvas.height - proj.position.y, Tuple::color(1, 1, 1, 1));
            canvas.WritePixel(proj.position.x, canvas.height - proj.position.y+1, Tuple::color(1, 1, 1, 1));
            canvas.WritePixel(proj.position.x, canvas.height - proj.position.y-1, Tuple::color(1, 1, 1, 1));
        }
        catch(std::invalid_argument const& ex){
            std::cout << ex.what() << std::endl;
        };

    }
    canvas.ToPPMFile("../../canvas");

}

void challenge_clock(){
    int size = 1000;
    Canvas canvas(size, size);
    Matrix canvas_transform = Transformation::translation(size/2, size/2, 0);

    int numVertices = 12;
    Tuple p = Tuple::point(0, 0, 0);
    Matrix T = Transformation::translation(1, 0, 0);
    // Matrix R = Transformation::rotation_z(2.f * (355.f/113.f) / numVertices);
    Matrix R = Transformation::rotation_z(2.f * (HIP_PI) / numVertices);
    Matrix S = Transformation::scaling(size/2 - size/10);

    p = T * p;

    for (int i = 0; i < numVertices; ++i){
        p = R * p;
        Tuple Sp = S * p;
        std::cout << "Point Pos: " << Sp << std::endl;
        Tuple cp = canvas_transform * Sp;
        try{
            canvas.WritePixel(cp.x, cp.y, Tuple::color(1, 1, 1, 1));
        }
        catch(std::invalid_argument const& ex){
            std::cout << ex.what() << std::endl;
        };

    }
    canvas.ToPPMFile("../../canvas");

}

void challenge_ray_to_sphere(){
    float backdrop_z = 10.f;
    float backdrop_size = 7.f;
    float backdrop_half_size = backdrop_size / 2.f;
    int canvas_size = 64;
    float pixel_size = (float) backdrop_size / (float) canvas_size;
    Tuple color = Tuple::color(1, 0, 0, 1);
    Canvas canvas(canvas_size, canvas_size);

    Sphere s;
    Matrix shear = Transformation::shearing(1, 0, 0, 0, 0, 0);
    Matrix rot = Transformation::rotation_y((355.f/113.f) / 2.f);
    Matrix scale = Transformation::scaling(0.75f);
    s.transformation = scale * rot * shear;

    Tuple origin = Tuple::point(0, 0, -5);
    Ray r(origin, Tuple::vector(0, 0, 0));

    for (int y = 0; y < canvas_size; ++y) {
        float world_y = backdrop_half_size - ( pixel_size * y );
        for (int x = 0; x < canvas_size; ++x) {
            float world_x = -backdrop_half_size + ( pixel_size * x );
            Tuple target = Tuple::point(world_x, world_y, backdrop_z);
            r.direction = Tuple::normalize(target - r.origin);
            std::vector<Intersection> xs = Intersection::Intersect(s, r);
            Intersection* h = Intersection::Hit(xs);
            if (h) {
                try { canvas.WritePixel(x, y, color); }
                catch (std::invalid_argument const &ex) { std::cout << ex.what() << std::endl; };
            }
        }
    }
    canvas.ToPPMFile("../../canvas");
}

__global__ void render_kernel(Canvas canvas, Sphere s, Ray r, float backdrop_z, float backdrop_half_size, float pixel_size, Tuple color){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int canvas_size = canvas.width;

    if (x < canvas_size && y < canvas_size) {
        float world_y = backdrop_half_size - (pixel_size * y);
        float world_x = -backdrop_half_size + (pixel_size * x);
//        Tuple ToTarget = Tuple::vector(world_x - r.origin.x, world_y - r.origin.y, backdrop_z - r.origin.z);
        Tuple target = Tuple::point(world_x, world_y, backdrop_z);
        r.direction = Tuple::normalize( target - r.origin);

        std::vector<Intersection> xs = Intersection::Intersect(s, r);
        Intersection *h = Intersection::Hit(xs);
        if (h) {
            canvas.WritePixel(x, y, color);
        }
    }
}

void challenge_ray_to_sphere_CUDA(){
    float backdrop_z = 10.f;
    float backdrop_size = 7.f;
    float backdrop_half_size = backdrop_size / 2.f;
    int canvas_size = 64;
    float pixel_size = (float) backdrop_size / (float) canvas_size;
    Tuple color = Tuple::color(1, 0, 0, 1);
    Canvas canvas(canvas_size, canvas_size);

    Sphere s;
    Matrix shear = Transformation::shearing(1, 0, 0, 0, 0, 0);
    Matrix rot = Transformation::rotation_y((355.f/113.f) / 2.f);
    Matrix scale = Transformation::scaling(0.75f);
    s.transformation = scale * rot * shear;

    Tuple origin = Tuple::point(0, 0, -5);
    Ray r(origin, Tuple::vector(0, 0, 0));

    // Define CUDA grid and block sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((canvas_size + blockSize.x - 1) / blockSize.x, (canvas_size + blockSize.y - 1) / blockSize.y);

    Canvas* d_canvas;
    Sphere* d_sphere;
    Tuple* d_ray;
    Tuple* d_color;

    hipMalloc(&d_canvas, sizeof (Canvas));
    hipMalloc(&d_sphere, sizeof (Sphere));
    hipMalloc(&d_ray, sizeof (Ray));
    hipMalloc(&d_color, sizeof (Tuple));

    hipMemcpy(d_canvas, &canvas, sizeof (Canvas), hipMemcpyHostToDevice);
    hipMemcpy(d_sphere, &s, sizeof (Sphere), hipMemcpyHostToDevice);
    hipMemcpy(d_ray, &r, sizeof (Ray), hipMemcpyHostToDevice);
    hipMemcpy(d_color, &color, sizeof (Tuple), hipMemcpyHostToDevice);

    // Launch the kernel
    render_kernel<<<gridSize, blockSize>>>(canvas, s, r, backdrop_z, backdrop_half_size, pixel_size, color);

    // Copy the results back to host memory
    hipMemcpy(&canvas, d_canvas, sizeof (Canvas), hipMemcpyDeviceToHost);

    hipFree(d_canvas);
    hipFree(d_sphere);
    hipFree(d_ray);
    hipFree(d_color);

    canvas.ToPPMFile("../../canvas");
}

int main()
{
//    challenge_clock();
    challenge_ray_to_sphere();
    return 0;
}