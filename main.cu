#include <iostream>
#include "Tuple.h"
#include "Canvas.h"
#include "Matrix.h"
#include "Transformation.h"
#include "Sphere.h"
#include "Intersection.h"
#include <filesystem>
#include <cmath>

#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>

struct Projectile
{
    Tuple position;
    Tuple velocity;
    Projectile(const Tuple &pos, const Tuple &vel) : position(pos), velocity(vel) {}
};

struct Environment
{
    Tuple gravity;
    Tuple wind;
    Environment(const Tuple &gravity, const Tuple &wind) : gravity(gravity), wind(wind) {}
};

void tick(Projectile* proj, Environment* env) {
    proj->position += proj->velocity;
    proj->velocity += env->gravity + env->wind;
}

void challenge_projectile(){
    Canvas canvas(900, 550);

    Tuple initialPosition = Tuple::vector(0, 1, 0);
    Tuple initialVelocity = Tuple::normalize(Tuple::vector(1, 1.8, 0)) * 11.25;
    Projectile proj(initialPosition, initialVelocity);

    Tuple gravity = Tuple::vector(0, -0.1, 0);
    Tuple wind = Tuple::vector(-0.01, 0, 0);
    Environment env(gravity, wind );


    while (proj.position.y >= 0){
        tick(&proj, &env);
        std::cout << "Projectile Pos: " << proj.position << " Projectile Vel: " << proj.velocity << std::endl;
        try{
            canvas.WritePixel(proj.position.x, canvas.height - proj.position.y, Tuple::color(1, 1, 1, 1));
            canvas.WritePixel(proj.position.x+1, canvas.height - proj.position.y, Tuple::color(1, 1, 1, 1));
            canvas.WritePixel(proj.position.x-1, canvas.height - proj.position.y, Tuple::color(1, 1, 1, 1));
            canvas.WritePixel(proj.position.x, canvas.height - proj.position.y+1, Tuple::color(1, 1, 1, 1));
            canvas.WritePixel(proj.position.x, canvas.height - proj.position.y-1, Tuple::color(1, 1, 1, 1));
        }
        catch(std::invalid_argument const& ex){
            std::cout << ex.what() << std::endl;
        };

    }
    canvas.ToPPMFile("../../canvas");

}

void challenge_clock(){
    int size = 1000;
    Canvas canvas(size, size);
    Matrix canvas_transform = Transformation::translation(size/2, size/2, 0);

    int numVertices = 12;
    Tuple p = Tuple::point(0, 0, 0);
    Matrix T = Transformation::translation(1, 0, 0);
    // Matrix R = Transformation::rotation_z(2.f * (355.f/113.f) / numVertices);
    Matrix R = Transformation::rotation_z(2.f * (HIP_PI) / numVertices);
    Matrix S = Transformation::scaling(size/2 - size/10);

    p = T * p;

    for (int i = 0; i < numVertices; ++i){
        p = R * p;
        Tuple Sp = S * p;
        std::cout << "Point Pos: " << Sp << std::endl;
        Tuple cp = canvas_transform * Sp;
        try{
            canvas.WritePixel(cp.x, cp.y, Tuple::color(1, 1, 1, 1));
        }
        catch(std::invalid_argument const& ex){
            std::cout << ex.what() << std::endl;
        };

    }
    canvas.ToPPMFile("../../canvas");

}

void challenge_ray_to_sphere(){
    float backdrop_z = 10.f;
    float backdrop_size = 7.f;
    float backdrop_half_size = backdrop_size / 2.f;
    int canvas_size = 64;
    float pixel_size = (float) backdrop_size / (float) canvas_size;
    Tuple color = Tuple::color(1, 0, 0, 1);
    Canvas canvas(canvas_size, canvas_size);

    Sphere s;
    Matrix shear = Transformation::shearing(1, 0, 0, 0, 0, 0);
    Matrix rot = Transformation::rotation_y((355.f/113.f) / 2.f);
    Matrix scale = Transformation::scaling(0.75f);
    s.transformation = scale * rot * shear;

    Tuple origin = Tuple::point(0, 0, -5);
    Ray r(origin, Tuple::vector(0, 0, 0));

    for (int y = 0; y < canvas_size; ++y) {
        float world_y = backdrop_half_size - ( pixel_size * y );
        for (int x = 0; x < canvas_size; ++x) {
            float world_x = -backdrop_half_size + ( pixel_size * x );
            Tuple target = Tuple::point(world_x, world_y, backdrop_z);
            r.direction = Tuple::normalize(target - r.origin);
            std::vector<Intersection> xs = Intersection::Intersect(s, r);
            Intersection* h = Intersection::Hit(xs);
            if (h) {
                try { canvas.WritePixel(x, y, color); }
                catch (std::invalid_argument const &ex) { std::cout << ex.what() << std::endl; };
            }
        }
    }
    canvas.ToPPMFile("../../canvas");
}

int main()
{
//    challenge_clock();
    challenge_ray_to_sphere();
    return 0;
}